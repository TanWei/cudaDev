#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h>
#include <stdio.h>

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

__global__ void add(const double* x, const double* y, double* z, int N);

void check(const double* z, const int N);

//#define CHECK(call) call

int main()
{
    const int N = 100000000;
    const int M = sizeof(double) * N;
    double* h_x = (double*) malloc(M);
    double* h_y = (double*) malloc(M);
    double* h_z = (double*) malloc(M);
    for (int n=0; n < N; n++)
    {
        h_x[n] = a;
        h_y[n] = b;
    }
    double *d_x, *d_y, *d_z;
    CHECK(hipMalloc((void**)&d_x, M));
    CHECK(hipMalloc((void**)&d_y, M));
    CHECK(hipMalloc((void**)&d_z, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));
    const int block_size = 128;
    const int grid_size = N / block_size;
    printf("block_size:%d, grid_size:%d\n", block_size, grid_size);

    //add<<<grid_size, block_size>>>(d_x, d_y, d_z, N); 
    add<<<grid_size, block_size>>>(d_x, d_y, d_z, N);
    CHECK(hipGetLastError()); //捕捉下面一句话之前最后一个错误
    CHECK(hipDeviceSynchronize()); //主机和设备同步

    CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));
    check(h_z, N);

    free(h_x);
    free(h_y);
    free(h_z);
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));
    return 0;
}

__global__ void add(const double* x, const double* y, double* z, int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n >= N )
    {
        return;
    }
    z[n] = x[n] + y[n];
    if (n < 10)
    {
        printf("from block:%d, thread:%d\n", blockIdx.x, threadIdx.x);
    }
    if (n==10)
    {
        printf("========================");
    }
    if (n > N - 10)
    {
        printf("from block:%d, thread:%d\n", blockIdx.x, threadIdx.x);
    }
}

void check(const double* z, const int N)
{
    bool has_error = false;
    for (int n=0; n<N; n++)
    {
        if (fabs(z[n] - c) > EPSILON)
        {
            has_error = true;
            break;
        }
    }
    printf("%s", has_error ? "has errot" : "no error");
}
