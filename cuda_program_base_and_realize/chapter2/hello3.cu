
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf("hello from gpu\n");
}

int main()
{
    hello_from_gpu<<<2, 4>>>();
    hipDeviceSynchronize();
	return 0;
}
