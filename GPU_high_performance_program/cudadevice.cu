#include "../common/book.h"

int main (void)
{
    hipDeviceProp_t prop;
    int cout;
    HANDLE_ERROR( hipGetDeviceCount(&count));
    for (int i=0; i<count; i++)
    {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
        
    }

    hipDeviceProp_t prop_my;
    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop_my.major = 1; // 设备计算功能集的主版本号
    prop.minor = 3; //设备计算功能集的次版本号
    HANDLE_ERROR( hipChooseDevice(&dev, &prop) );
    prinf("fu he de gpu id is: %d", dev);
    HANDLE_ERROR( hipSetDevice(dev) );
}